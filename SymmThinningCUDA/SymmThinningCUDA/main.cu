#include <vector>
#include <iostream>
#include <cstdlib>

#include "thinning.h"
#include "h5_io.h"

// For profiling execution times
#include <chrono>
#ifndef TIMER_END
#define TIMER_END(str, start) std::cout << std::setw(6) << std::right << \
  std::chrono::duration_cast<std::chrono::milliseconds>( \
	std::chrono::high_resolution_clock::now()-start).count() << \
	" ms " << str << std::endl;
#endif

int main(int argc, char *argv[])
{

	if (argc < 4) {
		std::cout << "arguments: depth directory maxIter p (e.g. 256 oldH5 80 10)" << std::endl;
		return 1;
	}
	unsigned numSlices = std::atoi(argv[1]);
	unsigned width = numSlices, height = numSlices;
	unsigned maxIter = std::atoi(argv[3]);
	unsigned p = std::atoi(argv[4]);

	std::cout << "using dimension:" << std::endl;
	std::cout << "  numSlices:" << numSlices << std::endl;
	std::cout << "      width:" << width << std::endl;
	std::cout << "     height:" << height << std::endl;
	std::cout << "initial data from:" << std::endl;
	std::cout << "   " << argv[2] << std::endl;
	std::cout << "other configuration elements:" << std::endl;
	std::cout << "    maxIter:" << maxIter << std::endl;
	std::cout << "          p:" << p << std::endl;

	thin::IjkType size3D = thin::makeIjk(width, height, numSlices);
	unsigned maxNumVoxelsPerChunk = 100000U;

	h5_io::H5SliceIoManager sliceIoMngr("", argv[2], "newH5", width, height, numSlices, "chunkMap.txt", maxNumVoxelsPerChunk);

	thin::initDevice();
	thin::setNumThreadsPerBlock(196U);

	auto TIMER = std::chrono::high_resolution_clock::now();
	thin::chunkwiseThinning(sliceIoMngr, size3D, curIter, curDim, p, maxIter);
	TIMER_END("> main::chunkwiseThinning()", TIMER);
	thin::shutdownDevice();


	return 0;
}
